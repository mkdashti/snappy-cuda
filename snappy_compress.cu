#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>

#include "snappy_compress.h"


/**
 * This value could be halfed or quartered to save memory
 * at the cost of slightly worse compression.
 */
#define MAX_HASH_TABLE_BITS 14
#define MAX_HASH_TABLE_SIZE (1U << MAX_HASH_TABLE_BITS)

/**
 * Calculate the rounded down log base 2 of an unsigned integer.
 *
 * @param n: value to perform the calculation on
 * @return Log base 2 floor of n
 */
__host__ static inline int32_t log2_floor(uint32_t n)
{
	return (n == 0) ? -1 : 31 ^ __builtin_clz(n);
}
__device__ static inline int32_t log2_floor_d(uint32_t n)
{
	return (n == 0) ? -1 : 31 ^ __clz(n);
}

/**
 * Calculate the maximum expected compressed length for a given
 * uncompressed length.
 *	 
 * Compressed data can be defined as:
 *	  compressed := item* literal*
 *	  item		 := literal* copy
 *
 * The trailing literal sequence has a space blowup of at most 62/60
 * since a literal of length 60 needs one tag byte + one extra byte
 * for length information.
 *
 * Item blowup is trickier to measure.	Suppose the "copy" op copies
 * 4 bytes of data.  Because of a special check in the encoding code,
 * we produce a 4-byte copy only if the offset is < 65536.	Therefore
 * the copy op takes 3 bytes to encode, and this type of item leads
 * to at most the 62/60 blowup for representing literals.
 *
 * Suppose the "copy" op copies 5 bytes of data.  If the offset is big
 * enough, it will take 5 bytes to encode the copy op.	Therefore the
 * worst case here is a one-byte literal followed by a five-byte copy.
 * I.e., 6 bytes of input turn into 7 bytes of "compressed" data.
 *
 * This last factor dominates the blowup, so the final estimate is:
 */
__host__ __device__ static inline uint32_t snappy_max_compressed_length(uint32_t input_length) {
	if (input_length > 0) 
		return (32 + input_length + input_length / 6);
	else
		return 0;
}

/**
 * Write a varint to the output buffer. See the decompression code
 * for a description of this format.
 *
 * @param output: holds output buffer information
 * @param val: value to write
 */
__host__ __device__ static inline void write_varint32(struct host_buffer_context *output, uint32_t val)
{
	static const int mask = 128;

	if (val < (1 << 7)) {
		*(output->curr++) = val;
	}
	else if (val < (1 << 14)) {
		*(output->curr++) = val | mask;
		*(output->curr++) = val >> 7;
	}
	else if (val < (1 << 21)) {
		*(output->curr++) = val | mask;
		*(output->curr++) = (val >> 7) | mask;
		*(output->curr++) = val >> 14;
	}
	else if (val < (1 << 28)) {
		*(output->curr++) = val | mask;
		*(output->curr++) = (val >> 7) | mask;
		*(output->curr++) = (val >> 14) | mask;
		*(output->curr++) = val >> 21;
	}
	else {
		*(output->curr++) = val | mask;
		*(output->curr++) = (val >> 7) | mask;
		*(output->curr++) = (val >> 14) | mask;
		*(output->curr++) = (val >> 21) | mask;
		*(output->curr++) = val >> 28;
	}
}

/**
 * Write an unsigned integer to the output buffer.
 *
 * @param ptr: pointer where to write the integer
 * @param val: value to write
 */
__host__ __device__ static inline void write_uint32(uint8_t *ptr, uint32_t val)
{
	*(ptr++) = val & 0xFF;
	*(ptr++) = (val >> 8) & 0xFF;
	*(ptr++) = (val >> 16) & 0xFF;
	*(ptr++) = (val >> 24) & 0xFF;
}

/**
 * Read an unsigned integer from the input buffer.
 *
 * @param ptr: where to read the integer from
 * @return Value read
 */
__host__ __device__ static inline uint32_t read_uint32(uint8_t *ptr)
{
	uint32_t val = 0;
	
	val |= *ptr++ & 0xFF;
	val |= (*ptr++ & 0xFF) << 8;
	val |= (*ptr++ & 0xFF) << 16;
	val |= (*ptr++ & 0xFF) << 24;
	return val;
}

/**
 * Get the size of the hash table needed for the size we are
 * compressing, and reset the values in the table.
 *
 * @param table: pointer to the start of the hash table
 * @param size_to_compress: size we are compressing
 * @param table_size[out]: size of the table needed to compress size_to_compress
 */
__host__ __device__ static inline void get_hash_table(uint16_t *table, uint32_t size_to_compress, uint32_t *table_size)
{
	*table_size = 256;
	while ((*table_size < MAX_HASH_TABLE_SIZE) && (*table_size < size_to_compress))
		*table_size <<= 1;

	memset(table, 0, *table_size * sizeof(*table));
}

/**
 * Hash function.
 *
 * Any hash function will produce a valid compressed bitstream, but a good
 * hash function reduces the number of collisions and thus yields better
 * compression for compressible input, and more speed for incompressible
 * input. Of course, it doesn't hurt if the hash function is reasonably fast
 * either, as it gets called a lot.
 *
 * @param ptr: pointer to the value we want to hash
 * @param shift: adjusts hash to be within table size
 * @return Hash of four bytes stored at ptr
 */
__host__ __device__ static inline uint32_t hash(uint8_t *ptr, int shift)
{
	uint32_t kmul = 0x1e35a7bd;
	uint32_t bytes = read_uint32(ptr);
	return (bytes * kmul) >> shift;
}

/**
 * Find the number of bytes in common between s1 and s2.
 *
 * @param s1: first buffer to compare
 * @param s2: second buffer to compare
 * @param s2_limit: end of second buffer to compare
 * @return Number of bytes in common between s1 and s2
 */
__host__ __device__ static inline int32_t find_match_length(uint8_t *s1, uint8_t *s2, uint8_t *s2_limit)
{
	int32_t matched = 0;
	
	// Check by increments of 4 first
	while ((s2 <= (s2_limit - 4)) && (read_uint32(s2) == read_uint32(s1 + matched))) {
		s2 += 4;
		matched += 4;
	}

	// Remaining bytes
	while ((s2 < s2_limit) && (s1[matched] == *s2)) {
		s2++;
		matched++;
	}
	
	return matched;
}

/**
 * Emit a literal element.
 *
 * @param output: holds output buffer information
 * @param literal: buffer storing the literal data
 * @param len: length of the literal
 */
__host__ __device__ static void emit_literal(struct host_buffer_context *output, uint8_t *literal, uint32_t len)
{
	//printf("emit_literal %d %d\n", len, output->curr-output->buffer);
	uint32_t n = len - 1; // Zero-length literals are disallowed
	
	if (n < 60) {
		*output->curr++ = EL_TYPE_LITERAL | (n << 2);
	}
	else {
		uint8_t *base = output->curr;
		uint8_t count = 0;
		output->curr++;
		while (n > 0) {
			*output->curr++ = n & 0xFF;
			n >>= 8;
			count++;
		}

		*base = EL_TYPE_LITERAL | ((59 + count) << 2);
	}
	
	memcpy(output->curr, literal, len);
	output->curr += len;
}

/**
 * Emit a copy element that is less than 64-bytes in length.
 *
 * @param output: holds output buffer information
 * @param offset: offset of the copy
 * @param len: length of the copy
 */
__host__ __device__ static void emit_copy_less_than64(struct host_buffer_context *output, uint32_t offset, uint32_t len)
{
	if ((len < 12) && (offset < 2048)) {
		*output->curr++ = EL_TYPE_COPY_1 + ((len - 4) << 2) + ((offset >> 8) << 5);
		*output->curr++ = offset & 0xFF;
	}
	else {
		*output->curr++ = EL_TYPE_COPY_2 + ((len - 1) << 2);
		*output->curr++ = offset & 0xFF;
		*output->curr++ = (offset >> 8) & 0xFF;
	}
}

/**
 * Emit copy elements in chunks of length 64-bytes.
 *
 * @param output: holds output buffer information
 * @param offset: offset of the copy
 * @param len: length of the copy
 */
__host__ __device__ static void emit_copy(struct host_buffer_context *output, uint32_t offset, uint32_t len) 
{
	//printf("emit_copy %d %d %d\n", offset, len, output->curr - output->buffer);
	
	// Emit 64-byte copies but keep at least four bytes reserved
	while (len >= 68) {
		emit_copy_less_than64(output, offset, 64);
		len -= 64;
	}

	// Emit an extra 60-byte copy if we have too much data to fit in one copy
	if (len > 64) {
		emit_copy_less_than64(output, offset, 60);
		len -= 60;
	}

	// Emit remainder
	emit_copy_less_than64(output, offset, len);
}

/**
 * Perform Snappy compression on a block of input data, and save the compressed
 * data to the output buffer.
 *
 * @param input: holds input buffer information
 * @param output: holds output buffer information
 * @param input_size: size of the input to compress
 * @param table: pointer to allocated hash table
 * @param table_size: size of the hash table
 */
__host__ static void compress_block(struct host_buffer_context *input, struct host_buffer_context *output, uint32_t input_size, uint16_t *table, uint32_t table_size)
{
	uint8_t *base_input = input->curr;
	uint8_t *input_end = input->curr + input_size;
	const int32_t shift = 32 - log2_floor(table_size);

	// Make space for compressed length
	output->curr += 4;
	uint8_t *output_start = output->curr;

	/*
	 * Bytes in [next_emit, input->curr) will be emitted as literal bytes.
	 * Or [next_emit, input_end) after the main loop.
	 */
	uint8_t *next_emit = input->curr;
	const uint32_t input_margin_bytes = 15;

	if (input_size >= input_margin_bytes) {
		const uint8_t *const input_limit = input->curr + input_size - input_margin_bytes;
		
		uint32_t next_hash;
		for (next_hash = hash(++input->curr, shift);;) {
			/*
			 * The body of this loop calls EmitLiteral once and then EmitCopy one or
			 * more times.	(The exception is that when we're close to exhausting
			 * the input we goto emit_remainder.)
			 *
			 * In the first iteration of this loop we're just starting, so
			 * there's nothing to copy, so calling EmitLiteral once is
			 * necessary.  And we only start a new iteration when the
			 * current iteration has determined that a call to EmitLiteral will
			 * precede the next call to EmitCopy (if any).
			 *
			 * Step 1: Scan forward in the input looking for a 4-byte-long match.
			 * If we get close to exhausting the input then goto emit_remainder.
			 *
			 * Heuristic match skipping: If 32 bytes are scanned with no matches
			 * found, start looking only at every other byte. If 32 more bytes are
			 * scanned, look at every third byte, etc.. When a match is found,
			 * immediately go back to looking at every byte. This is a small loss
			 * (~5% performance, ~0.1% density) for lcompressible data due to more
			 * bookkeeping, but for non-compressible data (such as JPEG) it's a huge
			 * win since the compressor quickly "realizes" the data is incompressible
			 * and doesn't bother looking for matches everywhere.
			 *
			 * The "skip" variable keeps track of how many bytes there are since the
			 * last match; dividing it by 32 (ie. right-shifting by five) gives the
			 * number of bytes to move ahead for each iteration.
			 */
			uint32_t skip_bytes = 32;
			uint8_t *next_input = input->curr;
			uint8_t *candidate;
			do {
				input->curr = next_input;
				uint32_t hval = next_hash;
				uint32_t bytes_between_hash_lookups = skip_bytes++ >> 5;
				next_input = input->curr + bytes_between_hash_lookups;

				if (next_input > input_limit)
					goto emit_remainder;

				next_hash = hash(next_input, shift);
				candidate = base_input + table[hval];
				table[hval] = input->curr - base_input;
			} while (read_uint32(input->curr) != read_uint32(candidate));
			
			/*
			 * Step 2: A 4-byte match has been found.  We'll later see if more
			 * than 4 bytes match.	But, prior to the match, input bytes
			 * [next_emit, input->curr) are unmatched.	Emit them as "literal bytes."
			 */
			emit_literal(output, next_emit, input->curr - next_emit);

			/*
			 * Step 3: Call EmitCopy, and then see if another EmitCopy could
			 * be our next move.  Repeat until we find no match for the
			 * input immediately after what was consumed by the last EmitCopy call.
			 *
			 * If we exit this loop normally then we need to call EmitLiteral next,
			 * though we don't yet know how big the literal will be.  We handle that
			 * by proceeding to the next iteration of the main loop.  We also can exit
			 * this loop via goto if we get close to exhausting the input.
			 */
			uint8_t *insert_tail;
			uint32_t candidate_bytes = 0;

			do {
				/*
				 * We have a 4-byte match at input->curr, and no need to emit any
				 *	"literal bytes" prior to input->curr.
				 */
				const uint8_t *base = input->curr;
				int32_t matched = 4 + find_match_length(candidate + 4, input->curr + 4, input_end);
				input->curr += matched;

				int32_t offset = base - candidate;
				emit_copy(output, offset, matched);
			
				/*
				 * We could immediately start working at input->curr now, but to improve
				 * compression we first update table[Hash(input->curr - 1, ...)]/
				 */
				insert_tail = input->curr - 1;
				next_emit = input->curr;
				if (input->curr >= input_limit)
					goto emit_remainder;

				uint32_t prev_hash = hash(insert_tail, shift);
				table[prev_hash] = input->curr - base_input - 1;

				uint32_t curr_hash = hash(insert_tail + 1, shift);
				candidate = base_input + table[curr_hash];
				candidate_bytes = read_uint32(candidate);
				table[curr_hash] = input->curr - base_input;
			} while(read_uint32(insert_tail + 1) == candidate_bytes);

			next_hash = hash(insert_tail + 2, shift);
			input->curr++;
		}
	}
				
emit_remainder:
	/* Emit the remaining bytes as literal */
	if (next_emit < input_end) {
		emit_literal(output, next_emit, input_end - next_emit);
		input->curr = input_end;
	}

	write_uint32(output_start - 4, output->curr - output_start);
	//printf(" host compressed size? %d\n",*(output_start - 4));
}

/**
 * Perform Snappy compression on a block of input data, and save the compressed
 * data to the output buffer.
 *
 * @param input: holds input buffer information
 * @param output: holds output buffer information
 * @param input_size: size of the input to compress
 * @param table: pointer to allocated hash table
 * @param table_size: size of the hash table
 */
__device__ static void compress_block_d(struct host_buffer_context *input, struct host_buffer_context *output, uint32_t input_size, uint16_t *table, uint32_t table_size, uint32_t idx, uint32_t *output_offsets, uint32_t output_metadata_size)
{
	uint8_t *current_input = input->buffer+(idx * input->block_size);
	uint8_t *current_output = output->buffer+(idx * input->block_size);

	host_buffer_context temp_output;

	//printf("first character in block is %c\n", *current_input);

	uint8_t *base_input = current_input;
	uint8_t *input_end = current_input + input_size;
	const int32_t shift = 32 - log2_floor_d(table_size);


	if(idx == 0)
		current_output += output_metadata_size;

	// Make space for compressed length
	current_output += 4;
	temp_output.curr = current_output;
	uint8_t *output_start = current_output;

	/*
	 * Bytes in [next_emit, current_input) will be emitted as literal bytes.
	 * Or [next_emit, input_end) after the main loop.
	 */
	uint8_t *next_emit = current_input;
	const uint32_t input_margin_bytes = 15;

	if (input_size >= input_margin_bytes) {
		const uint8_t *const input_limit = current_input + input_size - input_margin_bytes;
			
		uint32_t next_hash;
		for (next_hash = hash(++current_input, shift);;) {
			/*
			 * The body of this loop calls EmitLiteral once and then EmitCopy one or
			 * more times.	(The exception is that when we're close to exhausting
			 * the input we goto emit_remainder.)
			 *
			 * In the first iteration of this loop we're just starting, so
			 * there's nothing to copy, so calling EmitLiteral once is
			 * necessary.  And we only start a new iteration when the
			 * current iteration has determined that a call to EmitLiteral will
			 * precede the next call to EmitCopy (if any).
			 *
			 * Step 1: Scan forward in the input looking for a 4-byte-long match.
			 * If we get close to exhausting the input then goto emit_remainder.
			 *
			 * Heuristic match skipping: If 32 bytes are scanned with no matches
			 * found, start looking only at every other byte. If 32 more bytes are
			 * scanned, look at every third byte, etc.. When a match is found,
			 * immediately go back to looking at every byte. This is a small loss
			 * (~5% performance, ~0.1% density) for lcompressible data due to more
			 * bookkeeping, but for non-compressible data (such as JPEG) it's a huge
			 * win since the compressor quickly "realizes" the data is incompressible
			 * and doesn't bother looking for matches everywhere.
			 *
			 * The "skip" variable keeps track of how many bytes there are since the
			 * last match; dividing it by 32 (ie. right-shifting by five) gives the
			 * number of bytes to move ahead for each iteration.
			 */
			uint32_t skip_bytes = 32;
			uint8_t *next_input = current_input;
			uint8_t *candidate;
			do {
				current_input = next_input;
				uint32_t hval = next_hash;
				uint32_t bytes_between_hash_lookups = skip_bytes++ >> 5;
				next_input = current_input + bytes_between_hash_lookups;

				if (next_input > input_limit)
					goto emit_remainder;

				next_hash = hash(next_input, shift);
				candidate = base_input + table[hval];
				table[hval] = current_input - base_input;
			} while (read_uint32(current_input) != read_uint32(candidate));
			
			/*
			 * Step 2: A 4-byte match has been found.  We'll later see if more
			 * than 4 bytes match.	But, prior to the match, input bytes
			 * [next_emit, current_input) are unmatched.	Emit them as "literal bytes."
			 */
			emit_literal(&temp_output, next_emit, current_input - next_emit);

			/*
			 * Step 3: Call EmitCopy, and then see if another EmitCopy could
			 * be our next move.  Repeat until we find no match for the
			 * input immediately after what was consumed by the last EmitCopy call.
			 *
			 * If we exit this loop normally then we need to call EmitLiteral next,
			 * though we don't yet know how big the literal will be.  We handle that
			 * by proceeding to the next iteration of the main loop.  We also can exit
			 * this loop via goto if we get close to exhausting the input.
			 */
			uint8_t *insert_tail;
			uint32_t candidate_bytes = 0;

			do {
				/*
				 * We have a 4-byte match at current_input, and no need to emit any
				 *	"literal bytes" prior to current_input.
				 */
				const uint8_t *base = current_input;
				int32_t matched = 4 + find_match_length(candidate + 4, current_input + 4, input_end);
				current_input += matched;

				int32_t offset = base - candidate;
				emit_copy(&temp_output, offset, matched);
			
				/*
				 * We could immediately start working at current_input now, but to improve
				 * compression we first update table[Hash(current_input - 1, ...)]/
				 */
				insert_tail = current_input - 1;
				next_emit = current_input;
				if (current_input >= input_limit)
					goto emit_remainder;

				uint32_t prev_hash = hash(insert_tail, shift);
				table[prev_hash] = current_input - base_input - 1;

				uint32_t curr_hash = hash(insert_tail + 1, shift);
				candidate = base_input + table[curr_hash];
				candidate_bytes = read_uint32(candidate);
				table[curr_hash] = current_input - base_input;
			} while(read_uint32(insert_tail + 1) == candidate_bytes);

			next_hash = hash(insert_tail + 2, shift);
			current_input++;
		}
	}
				
emit_remainder:
	/* Emit the remaining bytes as literal */
	if (next_emit < input_end) {
		emit_literal(&temp_output, next_emit, input_end - next_emit);
		current_input = input_end;
	}

	write_uint32(output_start - 4, temp_output.curr - output_start);
	//printf("compressed size? %d\n",*(output_start - 4));
	output_offsets[idx] = temp_output.curr - output_start + 4;

	
}

__global__ void snappy_compress_kernel(struct host_buffer_context *input, struct host_buffer_context *output, uint32_t *input_block_size_array, uint32_t total_blocks, uint32_t *output_offsets, uint32_t output_metadata_size, uint16_t **table)
{
    uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;

	if(idx < total_blocks)
	{
		// Get the size of the hash table used for this block
		uint32_t table_size;

		get_hash_table(table[idx], input_block_size_array[idx], &table_size);
		// Compress the current block
		compress_block_d(input, output, input_block_size_array[idx], table[idx], table_size, idx, output_offsets, output_metadata_size);

		
	}

}


/*************** Public Functions *******************/

void setup_compression(struct host_buffer_context *input, struct host_buffer_context *output, struct program_runtime *runtime) 
{
	struct timeval start;
	struct timeval end;
	gettimeofday(&start, NULL);

	/*
	 * Compressed data can be defined as:
	 *	  compressed := item* literal*
	 *	  item		 := literal* copy
	 *
	 * The trailing literal sequence has a space blowup of at most 62/60
	 * since a literal of length 60 needs one tag byte + one extra byte
	 * for length information.
	 *
	 * Item blowup is trickier to measure.	Suppose the "copy" op copies
	 * 4 bytes of data.  Because of a special check in the encoding code,
	 * we produce a 4-byte copy only if the offset is < 65536.	Therefore
	 * the copy op takes 3 bytes to encode, and this type of item leads
	 * to at most the 62/60 blowup for representing literals.
	 *
	 * Suppose the "copy" op copies 5 bytes of data.  If the offset is big
	 * enough, it will take 5 bytes to encode the copy op.	Therefore the
	 * worst case here is a one-byte literal followed by a five-byte copy.
	 * I.e., 6 bytes of input turn into 7 bytes of "compressed" data.
	 *
	 * This last factor dominates the blowup, so the final estimate is:
	 */
	uint32_t max_compressed_length = snappy_max_compressed_length(input->length);
	output->buffer = (uint8_t *)malloc(sizeof(uint8_t) * max_compressed_length);
	output->curr = output->buffer;
	output->length = 0;

	gettimeofday(&end, NULL);
	runtime->pre = get_runtime(&start, &end);
}
void setup_compression_cuda(struct host_buffer_context *input, struct host_buffer_context *output, struct program_runtime *runtime) 
{
	struct timeval start;
	struct timeval end;
	gettimeofday(&start, NULL);

	/*
	 * Compressed data can be defined as:
	 *	  compressed := item* literal*
	 *	  item		 := literal* copy
	 *
	 * The trailing literal sequence has a space blowup of at most 62/60
	 * since a literal of length 60 needs one tag byte + one extra byte
	 * for length information.
	 *
	 * Item blowup is trickier to measure.	Suppose the "copy" op copies
	 * 4 bytes of data.  Because of a special check in the encoding code,
	 * we produce a 4-byte copy only if the offset is < 65536.	Therefore
	 * the copy op takes 3 bytes to encode, and this type of item leads
	 * to at most the 62/60 blowup for representing literals.
	 *
	 * Suppose the "copy" op copies 5 bytes of data.  If the offset is big
	 * enough, it will take 5 bytes to encode the copy op.	Therefore the
	 * worst case here is a one-byte literal followed by a five-byte copy.
	 * I.e., 6 bytes of input turn into 7 bytes of "compressed" data.
	 *
	 * This last factor dominates the blowup, so the final estimate is:
	 */
	uint32_t max_compressed_length = snappy_max_compressed_length(input->length);
	//output->buffer = (uint8_t *)malloc(sizeof(uint8_t) * max_compressed_length);
	output->total_size = sizeof(uint8_t) * max_compressed_length;
	checkCudaErrors(hipMallocManaged(&output->buffer,output->total_size));
	output->curr = output->buffer;
	output->length = 0;

	gettimeofday(&end, NULL);
	runtime->pre = get_runtime(&start, &end);
}

snappy_status snappy_compress_host(struct host_buffer_context *input, struct host_buffer_context *output, uint32_t block_size)
{
	// Allocate the hash table for compression
	uint16_t *table = (uint16_t *)malloc(sizeof(uint16_t) * MAX_HASH_TABLE_SIZE);

	// Write the decompressed length
	uint32_t length_remain = input->length;
	write_varint32(output, length_remain);

	// Write the decompressed block size
	write_varint32(output, block_size);

	//while (input->curr < (input->buffer + input->length)) {
    while (length_remain > 0) {
		// Get the next block size ot compress
		uint32_t to_compress = MIN(length_remain, block_size);

		// Get the size of the hash table used for this block
		uint32_t table_size;
		get_hash_table(table, to_compress, &table_size);
		
		// Compress the current block
		compress_block(input, output, to_compress, table, table_size);
		
		length_remain -= to_compress;
	}

	// Update output length
	output->length = (output->curr - output->buffer);
	free(table);

	return SNAPPY_OK;
}

snappy_status snappy_compress_cuda(struct host_buffer_context *input, struct host_buffer_context *output, uint32_t block_size, struct program_runtime *runtime)
{
	// Write the decompressed length
	uint32_t length_remain = input->length;
	write_varint32(output, length_remain);

	// Write the decompressed block size
	write_varint32(output, block_size);

	uint32_t output_metadata_size = output->curr - output->buffer;

    uint32_t total_blocks = length_remain/block_size;
    uint32_t last_block_size = length_remain - (total_blocks * block_size);
    if(last_block_size)
        ++total_blocks;

    uint32_t *input_block_size_array = NULL;
	checkCudaErrors(hipMallocManaged(&input_block_size_array,sizeof(uint32_t) * total_blocks));
    for(int i = 0 ; i < total_blocks; i++)
        input_block_size_array[i] = block_size;
    if(last_block_size)
        input_block_size_array[total_blocks-1] = last_block_size;

	uint32_t *output_offsets;		//this will hold the end of each output portion for easy later merging
	checkCudaErrors(hipMallocManaged(&output_offsets,sizeof(uint32_t) * total_blocks));


	uint16_t **table;
	checkCudaErrors(hipMallocManaged(&table, sizeof(uint16_t *) * total_blocks));
	for(int i = 0; i < total_blocks; i++)
		checkCudaErrors(hipMallocManaged(&table[i], sizeof(uint16_t) * MAX_HASH_TABLE_SIZE));

	//CUDA calculation for grid and threads per block
	dim3 block(1);
	dim3 grid(total_blocks);
	if(runtime->blocks == 0 && runtime->threads_per_block == 0) //only set blocks and threads_per_block if user didn't set them
	{
		if (total_blocks >= 1024 * 1024 * 1024)
		{
			block.x = 512;
			grid.x = (unsigned int) ceil(total_blocks * 1.0 / block.x);
		}
	}
	else
	{
		grid.x = runtime->blocks;
		block.x = runtime->threads_per_block;
	}
	

	printf("---\nTotal blocks = %d\n", total_blocks);
	printf("block_size_array[last_block] = %d\n", input_block_size_array[total_blocks - 1]);
	printf("grid.x = %d , block.x = %d\n---\n", grid.x, block.x);

	int device = -1;
  	hipGetDevice(&device);
  	hipMemPrefetchAsync(output->buffer, output->total_size, device, NULL);
	hipMemPrefetchAsync(input->buffer, input->total_size, device, NULL);

    snappy_compress_kernel<<<grid,block,0>>>(input, output, input_block_size_array, total_blocks, output_offsets, output_metadata_size, table);
    checkCudaErrors(hipDeviceSynchronize());

	output->length += output_metadata_size;
	for(int i = 0; i < total_blocks; i++)
		output->length += output_offsets[i];



	// The first part of the output is the metadata (output_metadata_size bytes)
	// Every cuda thread will work on a block_size (32K) block independantly and write to its output block (also 32K)
	// We need to get first part (the meat!) of each output block and merge into the output buffer
	uint32_t length_so_far = output_metadata_size + output_offsets[0];
	for(int i = 1; i < total_blocks; i++) // no need to copy the first output block since it's in its right place
	{
		//printf("output->buffer %p output->buffer + metadata %p \n", output->buffer, output->buffer + output_metadata_size);
		//printf("size of copied compressed block %d \n", output_offsets[i]);
		
		memcpy(output->buffer + length_so_far, output->buffer+(i * input->block_size), sizeof(uint8_t) * output_offsets[i]);
		
		length_so_far += output_offsets[i];
	}

    checkCudaErrors(hipFree(input_block_size_array));
	checkCudaErrors(hipFree(output_offsets));
	for(int i = 0; i < total_blocks; i++)
		checkCudaErrors(hipFree(table[i]));
	checkCudaErrors(hipFree(table));

	return SNAPPY_OK;
}
